#include "hip/hip_runtime.h"
// [header]
// A very basic raytracer example.
// [/header]
// [compile]
// c++ -o raytracer -O3 -Wall raytracer.cpp
// [/compile]
// [ignore]
// Copyright (C) 2012  www.scratchapixel.com
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.
// [/ignore]
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include "cutil_math.h"


#if defined __linux__ || defined __APPLE__
  // "Compiled for Linux
#else
  // Windows doesn't define these values by default, Linux does
  #define M_PI 3.14159265359f  // pi
  #define INFINITY 1e8
#endif

#define width 1280  // screenwidth
#define height 1024 // screenheight
#define tileSize 16
#define MaxRayDepth 5 // This variable controls the maximum recursion depth

#define nbSpheres 100

// = Utils ===

inline float clamp (float x) { return x < 0.0f ? 0.0f : x > 1.0f ? 1.0f : x; }
inline int toInt (float x) { return int(clamp(x) * 255 + .5); }
inline float randF (float min, float max) { return min + (((float) rand()) / (float) RAND_MAX) * (max - min); }

// = Struct ===

struct Sphere {
  float3 center;                      /// position of the sphere
  // float radius, radius2;           /// sphere radius and radius^2
  float radius2;                      /// sphere radius^2
  float3 surfaceColor, emissionColor; /// surface color and emission (light)
  float transparency, reflection;     /// surface transparency and reflectivity

  // Compute a ray-sphere intersection using the geometric solution
  __device__ bool intersect (float3 rayorig, float3 raydir, float &t0, float &t1) const {
    float3 l = center - rayorig;
    float tca = dot(l, raydir);
    if (tca < 0) return false;
    float d2 = dot(l, l) - tca * tca;
    if (d2 > radius2) return false;
    float thc = sqrt(radius2 - d2);
    t0 = tca - thc;
    t1 = tca + thc;

    return true;
  }
};

__constant__ Sphere spheres[nbSpheres + 2];

__device__ float mix(const float &a, const float &b, const float &mix) {
  return b * mix + a * (1 - mix);
}

// This is the main trace function. It takes a ray as argument (defined by its origin
// and direction). We test if this ray intersects any of the geometry in the scene.
// If the ray intersects an object, we compute the intersection point, the normal
// at the intersection point, and shade this point using this information.
// Shading depends on the surface property (is it transparent, reflective, diffuse).
// The function returns a color for the ray. If the ray intersects an object that
// is the color of the object at the intersection point, otherwise it returns
// the background color.
__device__ float3 trace(
  const float3 rayorig,
  const float3 raydir,
  const int &depth
) {
  // if (raydir.length() != 1) std::cerr << "Error " << raydir << std::endl;
  float tnear = INFINITY;
  const Sphere* sphere = NULL;
  // find intersection of this ray with the sphere in the scene
  for (unsigned i = 0; i < nbSpheres + 2; ++i) {
    float t0 = INFINITY, t1 = INFINITY;
    if (spheres[i].intersect(rayorig, raydir, t0, t1)) {
      if (t0 < 0) t0 = t1;
      if (t0 < tnear) {
        tnear = t0;
        sphere = &spheres[i];
      }
    }
  }
  // if there's no intersection return black or background color
  if (!sphere) return make_float3(2);
  float3 surfaceColor = make_float3(0); // color of the ray/surfaceof the object intersected by the ray
  float3 phit = rayorig + raydir * tnear; // point of intersection
  float3 nhit = phit - sphere->center; // normal at the intersection point
  nhit = normalize(nhit); // normalize normal direction
  // If the normal and the view direction are not opposite to each other
  // reverse the normal direction. That also means we are inside the sphere so set
  // the inside bool to true. Finally reverse the sign of IdotN which we want
  // positive.
  float bias = 1e-4; // add some bias to the point from which we will be tracing
  bool inside = false;
  if (dot(raydir, nhit) > 0) nhit = -nhit, inside = true;

  if ((sphere->transparency > 0 || sphere->reflection > 0) && depth < MaxRayDepth) {
    float facingratio = -dot(raydir, nhit);
    // change the mix value to tweak the effect
    float fresneleffect = mix(pow(1 - facingratio, 3), 1, 0.1);
    // compute reflection direction (not need to normalize because all vectors
    // are already normalized)
    float3 refldir = raydir - nhit * 2 * dot(raydir, nhit);
    refldir = normalize(refldir);
    float3 reflection = trace(phit + nhit * bias, refldir, depth + 1);
    float3 refraction = make_float3(0);
    // if the sphere is also transparent compute refraction ray (transmission)
    if (sphere->transparency) {
      float ior = 1.1, eta = (inside) ? ior : 1 / ior; // are we inside or outside the surface?
      float cosi = -dot(nhit, raydir);
      float k = 1 - eta * eta * (1 - cosi * cosi);
      float3 refrdir = raydir * eta + nhit * (eta *  cosi - sqrt(k));
      refrdir = normalize(refrdir);
      refraction = trace(phit - nhit * bias, refrdir, depth + 1);
    }
    // the result is a mix of reflection and refraction (if the sphere is transparent)
    surfaceColor = (
      reflection * fresneleffect +
      refraction * (1 - fresneleffect) * sphere->transparency
    ) * sphere->surfaceColor;
  } else {
    // it's a diffuse object, no need to raytrace any further
    for (unsigned i = 0; i < nbSpheres + 2; ++i) {
      if (spheres[i].emissionColor.x > 0) {
        // this is a light
        float3 transmission = make_float3(1);
        float3 lightDirection = spheres[i].center - phit;
        lightDirection = normalize(lightDirection);
        for (unsigned j = 0; j < nbSpheres + 2; ++j) {
          if (i != j) {
            float t0, t1;
            if (spheres[j].intersect(phit + nhit * bias, lightDirection, t0, t1)) {
              transmission = make_float3(0);
              break;
            }
          }
        }
        surfaceColor +=
          sphere->surfaceColor * transmission *
          max(float(0), dot(nhit, lightDirection)) * spheres[i].emissionColor;
      }
    }
  }

  return surfaceColor + sphere->emissionColor;
}

__constant__ const float invWidth = 1 / float(width);
__constant__ const float invHeight = 1 / float(height);
__constant__ const float aspectratio = width / float(height);

__global__ void render_kernel (float3 *image) {
  float fov = 30.0f;
  float angle = tan(M_PI * 0.5 * fov / 180.0f);

  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int i = y * width + x;
  if (x > width || y > height) return;

  float xx = (2 * ((x + 0.5) * invWidth) - 1) * angle * aspectratio;
  float yy = (1 - 2 * ((y + 0.5) * invHeight)) * angle;
  float3 raydir = make_float3(xx, yy, -1);
  raydir = normalize(raydir);

  image[i] = trace(make_float3(0), raydir, 0);
}

// Main rendering function. We compute a camera ray for each pixel of the image
// trace it and return a color. If the ray hits a sphere, we return the color of the
// sphere at the intersection point, else we return the background color.
void render () {
  // Create image
  float3* image_h = new float3[width * height];
  float3* image_d;
  hipMalloc(&image_d, width * height * sizeof(float3));

  // Trace rays
  dim3 block(tileSize, tileSize, 1);
  dim3 grid(width / tileSize, height / tileSize, 1);
  render_kernel <<<grid, block>>> (image_d);
  hipMemcpy(image_h, image_d, width * height *sizeof(float3), hipMemcpyDeviceToHost);
  hipFree(image_d);

  // Save result to a PPM image (keep these flags if you compile under Windows)
  std::ofstream ofs("./untitled.ppm", std::ios::out | std::ios::binary);
  ofs << "P6\n" << width << " " << height << "\n255\n";
  for (unsigned i = 0; i < width * height; ++i) {
    ofs << (unsigned char) toInt(image_h[i].x) <<
           (unsigned char) toInt(image_h[i].y) <<
           (unsigned char) toInt(image_h[i].z);
  }
  ofs.close();
  delete [] image_h;
}

// In the main function, we will create the scene which is composed of 5 spheres
// and 1 light (which is also a sphere). Then, once the scene description is complete
// we render that scene, by calling the render() function.
int main(int argc, char **argv) {
  srand48(13);
  // Create scene on host
  Sphere *scene_h = new Sphere[nbSpheres + 2];
  // Spheres
  // float3 center, float radius2, float3 surfaceColor, float3 emissionColor, float transparency, float reflection
  scene_h[0] = { make_float3(0, -10004,  -2), 1e8, make_float3(0.2f), make_float3(0), 0, 0 }; // Background
  scene_h[1] = { make_float3(0,     20, -30),   9, make_float3(0.0f), make_float3(3), 0, 0 }; // Light
  for ( int i = 0; i < nbSpheres; ++i ) {

    float x,y,z,rd,r,b,g,t;
    x = (rand()/(1.*RAND_MAX))*20.-10.;
    y = (rand()/(1.*RAND_MAX))*2.-1.;
    z = (rand()/(1.*RAND_MAX))*10.-25.;
    rd = (rand()/(1.*RAND_MAX))*0.9+0.1;
    r  = (rand()/(1.*RAND_MAX));
    g  = (rand()/(1.*RAND_MAX));
    b  = (rand()/(1.*RAND_MAX));
    t  = (rand()/(1.*RAND_MAX))*0.5;
    scene_h[2 + i] = { make_float3(x, y, z), rd * rd, make_float3(r, g, b), make_float3(0), t, 1 };

    // float rd = randF(0.1f, 1.0f);
    // float tr = randF(0.5f, 1.0f);
    // float re = randF(0.0f, 1.0f);
    // float3 center = make_float3(randF(-10, 10), randF(-1, 1), randF(-25, -15));
    // float3 color = make_float3(randF(0, 1), randF(0, 1), randF(0, 1));
    // scene_h[2 + i] = { center, rd * rd, color, make_float3(0), tr, re };
  }
  // Copy the host's scene to a device constante
  hipMemcpyToSymbol(HIP_SYMBOL(spheres),  scene_h, (nbSpheres + 2) * sizeof(Sphere));
  delete[] scene_h;
  // Render the scene
  render();

  return 0;
}
